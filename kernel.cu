#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>
#include <map>
#include <stdio.h>
#include <float.h>

#include "glm/gtc/matrix_transform.hpp"

#include "Light.h"
#include "Camera.h"
#include "PointLight.h"
#include "Sphere.h"
#include "Triangle.h"
#include "glm/glm.hpp"
#include "PhongShader.h"
#include "CookTorranceShader.h"
#include "hipError_t.h"
#include "kernel.h"

#include "bvh.cpp"

#define kNoShapeFound NULL

const float kMaxDist = FLT_MAX;
using glm::vec3;
using glm::vec4;
using glm::mat4;
using std::vector;
using std::pair;

texture<uchar4, 2, hipReadModeNormalizedFloat> mytex;

// Only works with 24 bit images that are a power of 2
unsigned char* readBMP(char* filename, int *retWidth, int *retHeight)
{
   int i;
   FILE* f = fopen(filename, "rb");
   unsigned char info[54];
   fread(info, sizeof(unsigned char), 54, f); // read the 54-byte header

   // extract image height and width from header
   int width = *(int*)&info[18];
   int height = *(int*)&info[22];

   int size = 3 * width * height;
   unsigned char* data = new unsigned char[size]; // allocate 3 bytes per pixel
   unsigned char* retData = new unsigned char[size + width * height]; // allocate 4 bytes per pixel
   fread(data, sizeof(unsigned char), size, f); // read the rest of the data at once
   fclose(f);

   for(i = 0; i < width * height; i++)
   {
      retData[4 * i] = data[3 * i + 2];
      retData[4 * i + 1] = data[3 * i + 1];
      retData[4 * i + 2] = data[3 * i];
      retData[4 * i + 3] = 0;
   }

   delete data;
   *retWidth = width;
   *retHeight = height;
   return retData;
}

const bool kLeft = 0;
const bool kRight = 1;

typedef struct StackEntry {
   bool nextDir;
   BVHNode *node;
   __device__ StackEntry(BVHNode *stackNode = NULL, char nextDirection = 0) : node(stackNode), nextDir(nextDirection) {}
} StackEntry;

// Find the closest shape. The index of the intersecting object is stored in
// retOjIdx and the t-value along the input ray is stored in retParam
//
// toBeat can be set to a float value if you want to short-circuit as soon
// as you find an object closer than toBeat
//
// If no intersection is found, retObjIdx is set to 'kNoShapeFound'
__device__ void getClosestIntersection(const Ray &ray, BVHTree *tree, 
      Triangle **retObj, float *retParam, float toBeat = -FLT_MAX) {
   float t = kMaxDist;
   Triangle *closestGeom = kNoShapeFound;
   int maxDepth = 0;

   StackEntry stack[kMaxStackSize];
   int stackSize = 0;
   bool justPoppedStack = false;

   BVHNode *cursor = tree->root;
   bool nextDir;
     
   do {
      if (stackSize >= kMaxStackSize) {
         printf("Stack full, aborting!\n");
         return;
      }
         
      // If at a leaf
      if (cursor->geom) {
         maxDepth = max(maxDepth, stackSize);
         float dist = cursor->geom->getIntersection(ray);
         //If two shapes are overlapping, pick the one with the closest facing normal
         if (isFloatEqual(t, dist)) {
            glm::vec3 oldNorm = closestGeom->getNormalAt(ray, t);
            glm::vec3 newNorm = cursor->geom->getNormalAt(ray, dist);
            glm::vec3 eye = glm::normalize(-ray.d);
            float newDot = glm::dot(eye, newNorm);
            float oldDot = glm::dot(eye, oldNorm);
            if (newDot > oldDot) {
               closestGeom = cursor->geom;
               t = dist;
               if (t < toBeat) {
                  *retObj = closestGeom;
                  *retParam = t;
                  return;
               }
            }
         // Otherwise, if one face is front of the current one
         } else {
            if (dist < t && isFloatAboveZero(dist)) {
               t = dist;
               closestGeom = cursor->geom;
               if (t < toBeat) {
                  *retObj = closestGeom;
                  *retParam = t;
                  return;
               }
            }
         }
      // If not on a leaf and neither branch has been explored
      } else if (!justPoppedStack) { 
         if (!cursor->right) {
            cursor = cursor->left;
            justPoppedStack = false;
            continue;
         }

         // Go down the tree with the closest bounding box
         float right = cursor->right->bb.getIntersection(ray), left = cursor->left->bb.getIntersection(ray);
         if (isFloatAboveZero(right) && (right <= left || !isFloatAboveZero(left)) && right < t) {
            if (isFloatAboveZero(left)) stack[stackSize++] = StackEntry(cursor, kLeft);
            cursor = cursor->right;
            justPoppedStack = false;
            continue;
         } else if (isFloatAboveZero(left) && (left <= right || !isFloatAboveZero(right)) && left < t) {
            if (isFloatAboveZero(right)) stack[stackSize++] = StackEntry(cursor, kRight);
            cursor = cursor->left;
            justPoppedStack = false;
            continue;
         } 
      // If coming back from a 'recursion' and one of the branches hasn't been explored
      } else {
         if (nextDir == kRight) {
            float right = cursor->right->bb.getIntersection(ray);
            if (right < t) {
               cursor = cursor->right;
               justPoppedStack = false;
               continue;
            }
         } else {
            float left = cursor->left->bb.getIntersection(ray);
            if (left < t) {
               cursor = cursor->left;
               justPoppedStack = false;
               continue;
            }
         }
      }

      if(stackSize == 0) {
         break;
      }

      // Pop the stack
      cursor = stack[stackSize - 1].node; 
      nextDir = stack[stackSize - 1].nextDir;
      justPoppedStack = true;
      stackSize--;
   } while(true);

   *retObj = closestGeom;
   *retParam = t;
}

__device__ bool isInShadow(const Ray &shadow, BVHTree *tree, float intersectParam) {
   float closestIntersect;
   Triangle *closestObj;
   getClosestIntersection(shadow, tree, &closestObj, &closestIntersect, intersectParam);
   return isFloatLessThan(closestIntersect, intersectParam);
}

__device__ vec3 cosineWeightedSample(vec3 normal, float rand1, float rand2) {
   float distFromCenter = 1.0f - sqrt(rand1);
   float theta = 2.0f * M_PI * rand2;
   float phi = M_PI / 2.0f - acos(distFromCenter);

   float phiDeg = phi * 180.0f / M_PI;
   float thetaDeg = theta * 180.0f / M_PI;

   vec3 outV = normal.x < .99f ? glm::cross(normal, vec3(1.0f, 0.0, 0.0)) : vec3(0.0f, 1.0f, 0.0f); 
   glm::mat4 rot1 = glm::rotate(glm::mat4(1.0f), phiDeg, outV);
   glm::mat4 rot2 = glm::rotate(glm::mat4(1.0f), thetaDeg, normal);
   glm::vec4 norm(normal.x, normal.y, normal.z, 0.0f);
   
   return vec3(rot2 * rot1 * norm);
}


__device__ glm::vec3 getColor(Triangle *geom, Ray ray, float param) {
   Material m = geom->getMaterial();
   if (m.texId == NO_TEXTURE) {
      return m.clr;
   } else {
      glm::vec2 uv = geom->UVAt(ray, param);
      float4 clr = tex2D(mytex, uv.x, uv.y);
      return vec3(clr.x, clr.y, clr.z);
   }
}

//Note: The ray parameter must stay as a copy (not a reference) 
__device__ vec3 shadeObject(BVHTree *tree, 
      PointLight lights[], int g_lightCount, Triangle* geom, 
      float intParam, Ray ray, Shader **shader, hiprandState *randStates) {

   Material m = geom->getMaterial();
   if (isFloatZero(1.0f - m.refl - m.alpha)) return glm::vec3(0.0f); 

   glm::vec3 intersectPoint = ray.getPoint(intParam);
   vec3 normal = geom->getNormalAt(ray, intParam);
   vec3 matClr = getColor(geom, ray, intParam);
   vec3 eyeVec = glm::normalize(-ray.d);
   vec3 totalLight(0.0f);

   for(int lightIdx = 0; lightIdx < g_lightCount; lightIdx++) {
      vec3 light = lights[lightIdx].getLightAtPoint(intersectPoint);
      vec3 lightDir = lights[lightIdx].getLightDir(intersectPoint);
      Ray shadow = lights[lightIdx].getShadowFeeler(intersectPoint);
      float intersectParam = geom->getIntersection(shadow);
      bool inShadow = isInShadow(shadow, tree, intersectParam); 

      totalLight += (*shader)->shade(matClr, m.amb, m.dif, m.spec, m.rough, 
            eyeVec, lightDir, light, normal, 
            inShadow);
   }

   return totalLight * (1.0f - m.refl - m.alpha);
}

__global__ void initScene(Triangle geomList[], PointLight lights[],  TKTriangle *triangleTks, 
      int numTris, TKSmoothTriangle *smthTriTks, int numSmthTris, 
      TKPointLight *pLightTks, int numPointLights, Shader **shader, 
      ShadingType stype) {

   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   int gridSize = gridDim.x * blockDim.x;
   int geomListSize = 0;
   int lightListSize = 0;

   if (blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0 && threadIdx.y == 0) {
      // Setup the shader
      switch(stype) {
      case PHONG:
         *shader = new PhongShader(); 
         break;
      case COOK_TORRANCE:
         *shader = new CookTorranceShader();
         break;
      default:
         printf("Improper shading type specified\n");
         break;
      }
   }

   for (int triIdx = idx; triIdx < numTris; triIdx += gridSize) {
      const TKTriangle &t = triangleTks[triIdx];
      const TKFinish f = t.mod.fin;
      Material m(t.mod.pig.clr, t.mod.pig.f, f.amb, f.dif, f.spec, f.rough, f.refl, f.refr, f.ior, t.mod.pig.texId);
      geomList[triIdx + geomListSize] = Triangle(t.p1, t.p2, t.p3, t.n, t.n, t.n, m, t.vt1, t.vt2, t.vt3);
   }
   geomListSize += numTris;

   for (int smTriIdx = idx; smTriIdx < numSmthTris; smTriIdx += gridSize) {
      const TKSmoothTriangle &t = smthTriTks[smTriIdx];
      const TKFinish f = t.mod.fin;
      Material m(t.mod.pig.clr, t.mod.pig.f, f.amb, f.dif, f.spec, f.rough, f.refl, f.refr, f.ior, t.mod.pig.texId);
      geomList[smTriIdx + geomListSize] = Triangle(t.p1, t.p2, t.p3, t.n1, t.n2, t.n3, 
            m, t.vt1, t.vt2, t.vt3);

   }
   geomListSize += numSmthTris;

   for (int pointLightIdx = idx; pointLightIdx < numPointLights; pointLightIdx += gridSize) {
      TKPointLight &p = pLightTks[pointLightIdx];
      lights[pointLightIdx + lightListSize].p = p.pos; 
      lights[pointLightIdx + lightListSize].c = p.clr;
   }
   lightListSize += numPointLights;

}

__global__ void initCurand(hiprandState randStates[], int numRandStates) {
   int x = blockIdx.x * blockDim.x + threadIdx.x;

   if (x >= numRandStates) return;

   hiprand_init(x, 0, 0, &randStates[x]);
}

__global__ void generateCameraRays(int resWidth, int resHeight, Camera cam, RayCache cache[], hiprandState randStates[]) {
   int x = blockIdx.x * blockDim.x + threadIdx.x;
   int y = blockIdx.y * blockDim.y + threadIdx.y;
   float celWidth = 1.0f / (float)resWidth;
   float celHeight = 1.0f / (float)resHeight;

   if (x >= resWidth || y >= resWidth) return;

   int tid = threadIdx.y * blockDim.x + threadIdx.x;

   float uJitter = (hiprand_uniform(&randStates[tid]) - .5f) * celWidth;
   float vJitter = (hiprand_uniform(&randStates[tid]) - .5f) * celHeight;
   float u = ((float)(x - resWidth / 2) + .5f) * celWidth * 2.0f + uJitter; 
   float v = ((float)(y - resHeight / 2) + .5f) * celHeight * 2.0f + vJitter; 

   // .5f is because the magnitude of cam.right and cam.up should be equal
   // to the width and height of the image plane in world space
   vec3 rPos = u *.5f * cam.right + v * .5f * cam.up + cam.pos;
   vec3 rDir = rPos - cam.pos + cam.lookAtDir;
   int index = y * resWidth + x;
   cache[index] = RayCache(Ray(rPos, rDir), glm::vec3(1.0f));
}

__global__ void rayTrace(int column, int row, int resWidth, int resHeight,
      BVHTree *tree, PointLight lights[], int g_lightCount,  
      vec3 output[], Shader **shader, hiprandState randStates[], RayCache cache[], int depth) {

   __shared__ PointLight sLights[kMaxLights];
   int tid = threadIdx.y * blockDim.x + threadIdx.x; 
   int lightIdx = tid;
   while (lightIdx < g_lightCount) {
      sLights[lightIdx] = lights[lightIdx];
      lightIdx += blockDim.x * blockDim.y; 
   }
   __syncthreads();

   int x = blockIdx.x * blockDim.x + threadIdx.x + column;
   int y = blockIdx.y * blockDim.y + threadIdx.y + row;

   if (x >= resWidth || y >= resHeight) return;

   int cacheIdx = y * resWidth + x;
   Ray ray = cache[cacheIdx].ray;
   glm::vec3 scale = cache[cacheIdx].scale;

   vec3 totalColor(0.0f);
   bool killed = true;
   float t;
   Triangle *closestGeom;
   if (scale.x >= EPSILON || scale.y >= EPSILON || scale.z >= EPSILON) {
      getClosestIntersection(ray, tree, &closestGeom, &t);
      if (closestGeom != kNoShapeFound) {
         totalColor += scale * shadeObject(tree, sLights, g_lightCount, 
               closestGeom, t, ray, shader, &randStates[tid]);

         if (depth >= kMinDepth) {
            if (hiprand_uniform(&randStates[tid]) < kRussianRoulette) {
            } else {
               scale *= 1.0f / kRussianRoulette;
               killed = false;
            }
         } else {
            killed = false;
         }
      }
   } 

   if (!killed) {
      glm::vec3 normal = closestGeom->getNormalAt(ray, t);
      glm::vec3 point = ray.getPoint(t);
      float randNum = hiprand_uniform(&randStates[tid]); 
      Material m = closestGeom->getMaterial();
      float reflThresh = m.refl;
      float difThresh = 1.0f - m.alpha;

      Ray newRay(vec3(0.0f), vec3(0.0f));
      if (randNum < reflThresh) {
         vec3 eyeVec = glm::normalize(-ray.d);

         newRay = Ray(point, 2.0f * glm::dot(normal, eyeVec) * normal - eyeVec);

      // Shoot an indirect ray
      } else if (randNum < difThresh) {
         float rand1 = hiprand_uniform(&randStates[tid]); 
         float rand2 = hiprand_uniform(&randStates[tid]);
         vec3 dir = cosineWeightedSample(normal, rand1, rand2);
         newRay = Ray(point, dir);
         scale = m.dif * m.clr / (1.0f - m.alpha - m.refl);

      // Do refraction
      } else {
         vec3 eyeVec = glm::normalize(-ray.d);
         vec3 refrNorm;
         vec3 d = -eyeVec;
         float n1, n2;

         if (isFloatLessThan(glm::dot(eyeVec, normal), 0.0f)) {
            n1 = m.ior; n2 = kAirIOR;
            refrNorm = -normal;
         } else { 
            n1 = kAirIOR; n2 = m.ior;
            refrNorm = normal;
         }

         float dDotN = glm::dot(d, refrNorm);
         float nr = n1 / n2;
         float discriminant = 1.0f - nr * nr * (1.0f - dDotN * dDotN);
         if (discriminant > 0.0f) {
            vec3 refracDir = nr * (d - refrNorm * dDotN) - refrNorm * sqrtf(discriminant);
            newRay = Ray(point, refracDir);
         }
      }

      newRay.o += newRay.d * BIG_EPSILON;
      cache[cacheIdx] = RayCache(newRay, scale);
   }

   totalColor = vec3(clamp(totalColor.x, 0, 1), 
                         clamp(totalColor.y, 0, 1), 
                         clamp(totalColor.z, 0, 1)); 

   output[x + y * resWidth] += totalColor;
}

void allocateGPUScene(const TKSceneData &data, Triangle **dGeomList,
      PointLight **g_dLightList, int *retGeometryCount, 
      int *retLightCount, Shader **g_dShader, ShadingType stype) {
   int geometryCount = 0;
   int g_lightCount = 0;
   int biggestListSize = 0;

   int imgWidth, imgHeight;
   unsigned char *texData = readBMP("blitz.bmp", &imgWidth, &imgHeight);

   int imgSize = sizeof(uchar4) * imgWidth * imgHeight;
   hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);

   hipArray* cu_array;
   hipMallocArray(&cu_array, &channelDesc, imgWidth, imgHeight );

   //copy image to device array cu_array – used as texture mytex on device
   HANDLE_ERROR(hipMemcpyToArray(cu_array, 0, 0, texData, imgSize, hipMemcpyHostToDevice));
   // set texture parameters
   
   mytex.addressMode[0] = hipAddressModeWrap;
   mytex.addressMode[1] = hipAddressModeWrap;
   mytex.filterMode = hipFilterModeLinear;
   mytex.normalized = true; 

   // Bind the array to the texture
   HANDLE_ERROR(hipBindTextureToArray(mytex, cu_array, channelDesc));

   TKPointLight *dPointLightTokens = NULL;
   TKTriangle *dTriangleTokens = NULL;
   TKSmoothTriangle *dSmthTriTokens = NULL;

   int triangleCount = data.triangles.size();
   if (triangleCount > 0) {
      HANDLE_ERROR(hipMalloc(&dTriangleTokens, sizeof(TKTriangle) * triangleCount));
      HANDLE_ERROR(hipMemcpy(dTriangleTokens, &data.triangles[0], 
               sizeof(TKTriangle) * triangleCount, hipMemcpyHostToDevice));
      geometryCount += triangleCount;
      if (triangleCount > biggestListSize) biggestListSize = triangleCount;
   }

   int smoothTriangleCount = data.smoothTriangles.size();
   if (smoothTriangleCount > 0) {
      HANDLE_ERROR(hipMalloc(&dSmthTriTokens, 
               sizeof(TKSmoothTriangle) * smoothTriangleCount));
      HANDLE_ERROR(hipMemcpy(dSmthTriTokens, &data.smoothTriangles[0],
               sizeof(TKSmoothTriangle) * smoothTriangleCount, hipMemcpyHostToDevice));
      geometryCount += smoothTriangleCount;
      if (smoothTriangleCount > biggestListSize) biggestListSize = smoothTriangleCount;
   }

   int pointLightCount = data.pointLights.size();
   if (pointLightCount > 0) {
      HANDLE_ERROR(hipMalloc(&dPointLightTokens, 
               sizeof(TKPointLight) * pointLightCount));
      HANDLE_ERROR(hipMemcpy(dPointLightTokens, &data.pointLights[0],
               sizeof(TKPointLight) * pointLightCount, hipMemcpyHostToDevice));
      g_lightCount += pointLightCount;
      if (pointLightCount > biggestListSize) biggestListSize = pointLightCount;
   }

   HANDLE_ERROR(hipMalloc(dGeomList, sizeof(Triangle) * geometryCount));
   HANDLE_ERROR(hipMalloc(g_dLightList, sizeof(PointLight) * g_lightCount));

   int blockSize = kBlockWidth * kBlockWidth;
   int gridSize = (biggestListSize - 1) / blockSize + 1;
   // Fill up GeomList and LightList with actual objects on the GPU
   initScene<<<gridSize, blockSize>>>(*dGeomList, *g_dLightList, 
         dTriangleTokens, triangleCount, dSmthTriTokens, smoothTriangleCount, 
         dPointLightTokens, pointLightCount, g_dShader, stype);

   hipDeviceSynchronize();
   checkCUDAError("initScene failed");

   if (dTriangleTokens) HANDLE_ERROR(hipFree(dTriangleTokens));
   if (dSmthTriTokens) HANDLE_ERROR(hipFree(dSmthTriTokens));

   *retGeometryCount = geometryCount;
   *retLightCount = g_lightCount;
}

__global__ void convertToUchar4(int resWidth, int resHeight, vec3 vec3Clrs[], 
      uchar4 output[], int passes) {

   int x = blockIdx.x * blockDim.x + threadIdx.x; 
   // Flip the image
   int y = blockIdx.y * blockDim.y + threadIdx.y; 
   int flippedY = resHeight - y - 1; 

   if (x >= resWidth || y >= resHeight) return;

   int idx = y * resWidth + x;
   vec3 clr = vec3Clrs[idx];
   clr = clr * 255.0f / (float)passes;

   uchar4 convClr;
   convClr.x = clamp(clr.x, 0, 255); convClr.y = clamp(clr.y, 0, 255); 
   convClr.z = clamp(clr.z, 0, 255); convClr.w = 255;
   output[flippedY * resWidth + x] = convClr;
}

RayCache *g_dRayCache;
BVHTree *g_dBvhTree;
hiprandState *g_dRandStates;
vec3 *g_dVec3Out;
Shader **g_dShader;
PointLight *g_dLightList;
int g_lightCount;
Camera g_camera(vec3(0.0f), vec3(0.0f), vec3(0.0f), vec3(0.0f));

extern "C" void init_kernel(const TKSceneData &data, ShadingType stype, int width, int height) {
   Triangle *dGeomList; 
   int geometryCount;

   dim3 dimBlock, dimGrid;

   HANDLE_ERROR(hipDeviceSetLimit(hipLimitMallocHeapSize, kGimmeLotsOfMemory));

   TKCamera camTK = *data.camera;
   g_camera = Camera(camTK.pos, camTK.up, camTK.right, 
                 glm::normalize(camTK.lookAt - camTK.pos));

   // Fill the geomList and light list with objects dynamically created on the GPU
   HANDLE_ERROR(hipMalloc(&g_dShader, sizeof(Shader*)));
   HANDLE_ERROR(hipMalloc(&g_dVec3Out , sizeof(vec3) * width * height));
   HANDLE_ERROR(hipMemset(g_dVec3Out, 0, sizeof(vec3) * width * height));

   allocateGPUScene(data, &dGeomList, &g_dLightList, &geometryCount, &g_lightCount, g_dShader, stype);

   hipDeviceSynchronize();
   checkCUDAError("AllocateGPUScene failed");

   HANDLE_ERROR(hipMalloc(&g_dBvhTree, sizeof(BVHTree)));
   formBVH(dGeomList, geometryCount, g_dBvhTree);

   HANDLE_ERROR(hipMalloc(&g_dRandStates, sizeof(hiprandState) * kBlockWidth * kBlockWidth));

   dimBlock = dim3(kBlockWidth * kBlockWidth);
   initCurand<<<1, dimBlock>>>(g_dRandStates, kBlockWidth * kBlockWidth);
   HANDLE_ERROR(hipMalloc(&g_dRayCache, sizeof(RayCache) * width* height));

}

extern "C" void launch_kernel(int width, int height, int maxDepth, int pass, uchar4 *dOutput) {

   dim3 dimBlock = dim3(kBlockWidth, kBlockWidth);
   dim3 camGrid((width - 1) / kBlockWidth + 1, (height - 1) / kBlockWidth + 1);
   dim3 dimGrid = dim3((kColumnsPerKernel - 1) / kBlockWidth + 1, (kColumnsPerKernel - 1) / kBlockWidth + 1);

   generateCameraRays<<<camGrid, dimBlock>>>(width, height, g_camera, g_dRayCache, g_dRandStates);
   if (pass == 1) hipMemset(g_dVec3Out, 0, sizeof(vec3) * width * height);

   hipDeviceSynchronize();
   for (int depth = 0; depth < maxDepth; depth++) {
      for (int x = 0; x < width ; x += kColumnsPerKernel) {
         for (int y = 0; y < height; y += kRowsPerKernel) {
            rayTrace<<<dimGrid, dimBlock>>>(x, y, width, height,
               g_dBvhTree, g_dLightList, g_lightCount, g_dVec3Out, 
               g_dShader, g_dRandStates, g_dRayCache, depth);
         }
      }
   }
   hipDeviceSynchronize();
   checkCUDAError("rayTrace kernel failed");

   dimBlock = dim3(kBlockWidth, kBlockWidth);
   dimGrid = dim3((width - 1) / kBlockWidth + 1, (height - 1) / kBlockWidth + 1);
   convertToUchar4<<<dimGrid, dimBlock>>>(width, height, g_dVec3Out, dOutput, pass); 

   hipDeviceSynchronize();
   checkCUDAError("convertToUchar4 kernel failed");
}

void translateCamera(glm::vec3 dir) {
   g_camera.pos += dir;
}

void rotateCameraSideways(float angle) {
   mat4 rot = glm::rotate(mat4(1.0f), angle, g_camera.up);
   g_camera.right = vec3(rot * vec4(g_camera.right.x, 
                                    g_camera.right.y, 
                                    g_camera.right.z, 0.0f));
   g_camera.lookAtDir = vec3(rot * vec4(g_camera.lookAtDir.x, 
                                    g_camera.lookAtDir.y, 
                                    g_camera.lookAtDir.z, 0.0f));
}
void rotateCameraVertically(float angle) {
   glm::mat4 rot = glm::rotate(glm::mat4(1.0f), angle, g_camera.right);
   g_camera.up = vec3(rot * vec4(g_camera.up.x, 
                                    g_camera.up.y, 
                                    g_camera.up.z, 0.0f));
   g_camera.lookAtDir = vec3(rot * vec4(g_camera.lookAtDir.x, 
                                    g_camera.lookAtDir.y, 
                                    g_camera.lookAtDir.z, 0.0f));
}

void kernelGetImage(uchar4 *dImage, uchar4 *hostImage, int width, int height) {
   HANDLE_ERROR(hipMemcpy(hostImage, dImage, 
            sizeof(uchar4) * width * height, hipMemcpyDeviceToHost));
   hipDeviceSynchronize();
}
