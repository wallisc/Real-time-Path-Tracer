#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>
#include <string>
#include <map>
#include <stdio.h>
#include <float.h>

#include "glm/gtc/matrix_transform.hpp"

#include "Camera.h"
#include "Sphere.h"
#include "Triangle.h"
#include "glm/glm.hpp"
#include "PhongShader.h"
#include "hipError_t.h"
#include "kernel.h"

#include "bvh.cpp"

#define kNoShapeFound NULL

const float kMaxDist = FLT_MAX;
using glm::vec3;
using glm::vec4;
using glm::mat4;
using std::vector;
using std::map;
using std::string;
using std::pair;

typedef texture<uchar4, 2, hipReadModeNormalizedFloat> Tex;

Tex tex0;
Tex tex1;
Tex tex2;
Tex tex3;
Tex tex4;
Tex tex5;
Tex tex6;
Tex tex7;
Tex tex8;
Tex tex9;
Tex tex10;
Tex tex11;

__device__ Tex getCudaTexture(int i) {
   switch(i) {
   case 0: return tex0;
   case 1: return tex1;
   case 2: return tex2;
   case 3: return tex3;
   case 4: return tex4;
   case 5: return tex5;
   case 6: return tex6;
   case 7: return tex7;
   case 8: return tex8;
   case 9: return tex9;
   case 10: return tex10;
   case 11: return tex11;
   }
   printf("Texture not found, returning the first texture\n");
   return tex0;
}

__host__ Tex &getTexture(int i) {
   switch(i) {
   case 0: return tex0;
   case 1: return tex1;
   case 2: return tex2;
   case 3: return tex3;
   case 4: return tex4;
   case 5: return tex5;
   case 6: return tex6;
   case 7: return tex7;
   case 8: return tex8;
   case 9: return tex9;
   case 10: return tex10;
   case 11: return tex11;
   }
   printf("Texture not found, returning the first texture\n");
   return tex0;
}

// Only works with 24 bit images that are a power of 2
unsigned char* readBMP(const char* filename, int *retWidth, int *retHeight)
{
   int i;
   FILE* f = fopen(filename, "rb");
   unsigned char info[54];
   fread(info, sizeof(unsigned char), 54, f); // read the 54-byte header

   // extract image height and width from header
   int width = *(int*)&info[18];
   int height = *(int*)&info[22];

   int size = 3 * width * height;
   unsigned char* data = new unsigned char[size]; // allocate 3 bytes per pixel
   unsigned char* retData = new unsigned char[size + width * height]; // allocate 4 bytes per pixel
   fread(data, sizeof(unsigned char), size, f); // read the rest of the data at once
   fclose(f);

   for(i = 0; i < width * height; i++)
   {
      retData[4 * i] = data[3 * i + 2];
      retData[4 * i + 1] = data[3 * i + 1];
      retData[4 * i + 2] = data[3 * i];
      retData[4 * i + 3] = 0;
   }

   delete data;
   *retWidth = width;
   *retHeight = height;
   return retData;
}

const bool kLeft = 0;
const bool kRight = 1;

typedef struct StackEntry {
   bool nextDir;
   BVHNode *node;
   __device__ StackEntry(BVHNode *stackNode = NULL, char nextDirection = 0) : node(stackNode), nextDir(nextDirection) {}
} StackEntry;

// Find the closest shape. The index of the intersecting object is stored in
// retOjIdx and the t-value along the input ray is stored in retParam
//
// toBeat can be set to a float value if you want to short-circuit as soon
// as you find an object closer than toBeat
//
// If no intersection is found, retObjIdx is set to 'kNoShapeFound'
__device__ void getClosestIntersection(const Ray &ray, BVHTree *tree, 
      Triangle **retObj, float *retParam, float toBeat = -FLT_MAX) {
   float t = kMaxDist;
   Triangle *closestGeom = kNoShapeFound;
   int maxDepth = 0;

   StackEntry stack[kMaxStackSize];
   int stackSize = 0;
   bool justPoppedStack = false;

   BVHNode *cursor = tree->root;
   bool nextDir;
     
   do {
      if (stackSize >= kMaxStackSize) {
         printf("Stack full, aborting!\n");
         return;
      }
         
      // If at a leaf
      if (cursor->geom) {
         maxDepth = max(maxDepth, stackSize);
         float dist = cursor->geom->getIntersection(ray);
         //If two shapes are overlapping, pick the one with the closest facing normal
         if (isFloatEqual(t, dist)) {
            glm::vec3 oldNorm = closestGeom->getNormalAt(ray, t);
            glm::vec3 newNorm = cursor->geom->getNormalAt(ray, dist);
            glm::vec3 eye = glm::normalize(-ray.d);
            float newDot = glm::dot(eye, newNorm);
            float oldDot = glm::dot(eye, oldNorm);
            if (newDot > oldDot) {
               closestGeom = cursor->geom;
               t = dist;
               if (t < toBeat) {
                  *retObj = closestGeom;
                  *retParam = t;
                  return;
               }
            }
         // Otherwise, if one face is front of the current one
         } else {
            if (dist < t && isFloatAboveZero(dist)) {
               t = dist;
               closestGeom = cursor->geom;
               if (t < toBeat) {
                  *retObj = closestGeom;
                  *retParam = t;
                  return;
               }
            }
         }
      // If not on a leaf and neither branch has been explored
      } else if (!justPoppedStack) { 
         float left = cursor->left->bb.getIntersection(ray);

         if (!cursor->right && isFloatAboveZero(left) && left < t) {
            cursor = cursor->left;
            justPoppedStack = false;
            continue;
         }

         // Go down the tree with the closest bounding box
         float right = cursor->right->bb.getIntersection(ray);

         if (isFloatAboveZero(right) && (right <= left || !isFloatAboveZero(left)) && right < t) {
            if (isFloatAboveZero(left)) stack[stackSize++] = StackEntry(cursor, kLeft);
            cursor = cursor->right;
            justPoppedStack = false;
            continue;
         } else if (isFloatAboveZero(left) && (left <= right || !isFloatAboveZero(right)) && left < t) {
            if (isFloatAboveZero(right)) stack[stackSize++] = StackEntry(cursor, kRight);
            cursor = cursor->left;
            justPoppedStack = false;
            continue;
         } 
      // If coming back from a 'recursion' and one of the branches hasn't been explored
      } else {
         if (nextDir == kRight) {
            float right = cursor->right->bb.getIntersection(ray);
            if (right < t) {
               cursor = cursor->right;
               justPoppedStack = false;
               continue;
            }
         } else {
            float left = cursor->left->bb.getIntersection(ray);
            if (left < t) {
               cursor = cursor->left;
               justPoppedStack = false;
               continue;
            }
         }
      }

      if(stackSize == 0) {
         break;
      }

      // Pop the stack
      cursor = stack[stackSize - 1].node; 
      nextDir = stack[stackSize - 1].nextDir;
      justPoppedStack = true;
      stackSize--;
   } while(true);

   *retObj = closestGeom;
   *retParam = t;
}

__device__ bool isInShadow(const Ray &shadow, BVHTree *tree, float intersectParam) {
   float closestIntersect;
   Triangle *closestObj;
   getClosestIntersection(shadow, tree, &closestObj, &closestIntersect, intersectParam);
   return isFloatLessThan(closestIntersect, intersectParam);
}

__device__ vec3 cosineWeightedSample(vec3 normal, float rand1, float rand2) {
   float distFromCenter = rand1;
   float theta = 2.0f * M_PI * rand2;
   float phi = M_PI / 2.0f - acos(distFromCenter);

   float phiDeg = phi * 180.0f / M_PI;
   float thetaDeg = theta * 180.0f / M_PI;

   vec3 outV; 
   if (normal.x > .99f) outV = vec3(0.0f, 1.0f, 0.0f);
   else if (normal.x < -.99f) outV = vec3(0.0f, -1.0f, 0.0f);
   else outV = glm::cross(normal, vec3(1.0f, 0.0, 0.0));
   glm::mat4 rot1 = glm::rotate(glm::mat4(1.0f), phiDeg, outV);
   glm::mat4 rot2 = glm::rotate(glm::mat4(1.0f), thetaDeg, normal);
   glm::vec4 norm(normal.x, normal.y, normal.z, 0.0f);
   
   return vec3(rot2 * rot1 * norm);
}


__device__ glm::vec3 getColor(Triangle *geom, Ray ray, float param) {
   Material m = geom->getMaterial();
   if (m.texId == NO_TEXTURE) {
      return m.clr;
   } else {
      glm::vec2 uv = geom->UVAt(ray, param);
      float4 clr = tex2D(getCudaTexture(m.texId), uv.x, uv.y);
      return vec3(clr.x, clr.y, clr.z);
   }
}

//Note: The ray parameter must stay as a copy (not a reference) 
__device__ vec3 shadeObject(BVHTree *tree, 
      Triangle *lights[], int lightCount, Triangle* geom, 
      float intParam, Ray ray, hiprandState *randStates) {

   Material m = geom->getMaterial();
   if (m.emissive) { return m.clr;}
   if (isFloatZero(1.0f - m.refl - m.alpha)) return glm::vec3(0.0f); 


   glm::vec3 intersectPoint = ray.getPoint(intParam);
   vec3 normal = geom->getNormalAt(ray, intParam);
   vec3 matClr = getColor(geom, ray, intParam);
   vec3 eyeVec = glm::normalize(-ray.d);
   vec3 totalLight(0.0f);

   for(int lightIdx = 0; lightIdx < lightCount; lightIdx++) {

      vec3 lightColor = lights[lightIdx]->mat.dif * lights[lightIdx]->mat.clr;

      // Randomly sample the area light with a random baryocentric coordinate
      float alpha = hiprand_uniform(randStates);
      float beta = hiprand_uniform(randStates) * (1.0f - alpha);
      vec3 lightPos = lights[lightIdx]->getPointFromBary(alpha, beta);

      vec3 lightDir = glm::normalize(lightPos - intersectPoint);
      Ray shadow(lightPos, -lightDir);
      shadow.o += shadow.d * BIG_EPSILON;
      float intersectParam = geom->getIntersection(shadow);
      bool inShadow = isInShadow(shadow, tree, intersectParam); 

      totalLight += PhongShader::shade(matClr, m.amb, m.dif, m.spec, m.rough, 
            eyeVec, lightDir, lightColor, normal, inShadow);
   }

   return totalLight * (1.0f - m.refl - m.alpha);
}

__global__ void initScene(Triangle geomList[], TKTriangle *triangleTks, 
      int numTris, TKSmoothTriangle *smthTriTks, int numSmthTris) {

   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   int gridSize = gridDim.x * blockDim.x;
   int geomListSize = 0;

   for (int triIdx = idx; triIdx < numTris; triIdx += gridSize) {
      const TKTriangle &t = triangleTks[triIdx];
      if (t.mod.fin.em) {
         Material m(t.mod.pig.clr, t.mod.fin.dif);
         geomList[triIdx + geomListSize] = Triangle(t.p1, t.p2, t.p3, t.n, t.n, t.n, m, t.mod.velocity, t.vt1, t.vt2, t.vt3);
      } else {  
         const TKFinish f = t.mod.fin;
         Material m(t.mod.pig.clr, t.mod.pig.f, f.amb, f.dif, f.spec, f.rough, f.refl, f.refr, f.ior, t.mod.pig.texId);
         geomList[triIdx + geomListSize] = Triangle(t.p1, t.p2, t.p3, t.n, t.n, t.n, m, t.mod.velocity, t.vt1, t.vt2, t.vt3);
      }

   }
   geomListSize += numTris;

   for (int smTriIdx = idx; smTriIdx < numSmthTris; smTriIdx += gridSize) {
      const TKSmoothTriangle &t = smthTriTks[smTriIdx];
      if (t.mod.fin.em) {
         Material m(t.mod.pig.clr, t.mod.fin.dif);
         geomList[smTriIdx + geomListSize] = Triangle(t.p1, t.p2, t.p3, t.n1, t.n2, t.n3, m, t.mod.velocity, t.vt1, t.vt2, t.vt3);
      } else {
         const TKFinish f = t.mod.fin;
         Material m(t.mod.pig.clr, t.mod.pig.f, f.amb, f.dif, f.spec, f.rough, f.refl, f.refr, f.ior, t.mod.pig.texId);
         geomList[smTriIdx + geomListSize] = Triangle(t.p1, t.p2, t.p3, t.n1, t.n2, t.n3, 
               m, t.mod.velocity, t.vt1, t.vt2, t.vt3);
      }
   }
   geomListSize += numSmthTris;
}

__global__ void setupLights(Triangle geomList[], int listSize, Triangle *lights[]) {
   int lightNum = 0;
   for (int i = 0; i < listSize; i++) {
      if (geomList[i].getMaterial().emissive) {
         lights[lightNum++] = geomList + i;
      }
   }
}

__global__ void initCurand(hiprandState randStates[], int numRandStates) {
   int x = blockIdx.x * blockDim.x + threadIdx.x;

   if (x >= numRandStates) return;

   hiprand_init(x, 0, 0, &randStates[x]);
}

__global__ void generateCameraRays(int resWidth, int resHeight, Camera cam, RayCache cache[], hiprandState randStates[]) {
   int x = blockIdx.x * blockDim.x + threadIdx.x;
   int y = blockIdx.y * blockDim.y + threadIdx.y;
   float celWidth = 1.0f / (float)resWidth;
   float celHeight = 1.0f / (float)resHeight;

   if (x >= resWidth || y >= resWidth) return;

   int tid = threadIdx.y * blockDim.x + threadIdx.x;

   float uJitter = (hiprand_uniform(&randStates[tid]) - .5f) * celWidth;
   float vJitter = (hiprand_uniform(&randStates[tid]) - .5f) * celHeight;
   float u = ((float)(x - resWidth / 2) + .5f) * celWidth * 2.0f + uJitter; 
   float v = ((float)(y - resHeight / 2) + .5f) * celHeight * 2.0f + vJitter; 

   // .5f is because the magnitude of cam.right and cam.up should be equal
   // to the width and height of the image plane in world space
   vec3 rPos = u *.5f * cam.right + v * .5f * cam.up + cam.pos;
   vec3 rDir = rPos - cam.pos + cam.lookAtDir;
   int index = y * resWidth + x;
   cache[index] = RayCache(Ray(rPos, rDir), glm::vec3(1.0f));
}

__global__ void rayTrace(int column, int row, int resWidth, int resHeight,
      BVHTree *tree, Triangle *lights[], int lightCount,  
      vec3 output[], hiprandState randStates[], RayCache cache[], int depth) {

   int tid = threadIdx.y * blockDim.x + threadIdx.x; 

   int x = blockIdx.x * blockDim.x + threadIdx.x + column;
   int y = blockIdx.y * blockDim.y + threadIdx.y + row;

   if (x >= resWidth || y >= resHeight) return;

   int cacheIdx = y * resWidth + x;
   Ray ray = cache[cacheIdx].ray;
   glm::vec3 scale = cache[cacheIdx].scale;

   vec3 totalColor(0.0f);
   bool killed = true;
   float t;
   Triangle *closestGeom;
   if (scale.x >= EPSILON || scale.y >= EPSILON || scale.z >= EPSILON) {
      getClosestIntersection(ray, tree, &closestGeom, &t);
      if (closestGeom != kNoShapeFound) {
         totalColor += scale * shadeObject(tree, lights, lightCount, 
               closestGeom, t, ray, &randStates[tid]);

         //killed = closestGeom->getMaterial().emissive || (depth >= kMinDepth && hiprand_uniform(&randStates[tid]) < kRussianRoulette);
         //if (!killed) scale *= 1.0f / kRussianRoulette;
         if (!closestGeom->getMaterial().emissive && depth >= kMinDepth) {
            if (hiprand_uniform(&randStates[tid]) > kRussianRoulette) {
               scale *= 1.0f / kRussianRoulette;
               killed = false;
            }
         } else if (!closestGeom->getMaterial().emissive) {
            killed = false;
         }
      }
   } 

   if (!killed) {
      glm::vec3 normal = closestGeom->getNormalAt(ray, t);
      glm::vec3 point = ray.getPoint(t);
      float randNum = hiprand_uniform(&randStates[tid]); 
      Material m = closestGeom->getMaterial();
      float reflThresh = m.refl;
      float difThresh = 1.0f - m.alpha;

      Ray newRay(vec3(0.0f), vec3(0.0f));
      if (randNum < reflThresh) {
         vec3 eyeVec = glm::normalize(-ray.d);

         newRay = Ray(point, 2.0f * glm::dot(normal, eyeVec) * normal - eyeVec);

      // Shoot an indirect ray
      } else if (randNum < difThresh) {
         float rand1 = hiprand_uniform(&randStates[tid]); 
         float rand2 = hiprand_uniform(&randStates[tid]);
         vec3 dir = cosineWeightedSample(normal, rand1, rand2);
         newRay = Ray(point, dir);
         scale = m.dif * m.clr / (1.0f - m.alpha - m.refl);

      // Do refraction
      } else {
         vec3 eyeVec = glm::normalize(-ray.d);
         vec3 refrNorm;
         vec3 d = -eyeVec;
         float n1, n2;

         if (isFloatLessThan(glm::dot(eyeVec, normal), 0.0f)) {
            n1 = m.ior; n2 = kAirIOR;
            refrNorm = -normal;
         } else { 
            n1 = kAirIOR; n2 = m.ior;
            refrNorm = normal;
         }

         float dDotN = glm::dot(d, refrNorm);
         float nr = n1 / n2;
         float discriminant = 1.0f - nr * nr * (1.0f - dDotN * dDotN);
         if (discriminant > 0.0f) {
            vec3 refracDir = nr * (d - refrNorm * dDotN) - refrNorm * sqrtf(discriminant);
            newRay = Ray(point, refracDir);
         }
      }

      newRay.o += newRay.d * BIG_EPSILON;
      cache[cacheIdx] = RayCache(newRay, scale);
   }

   totalColor = vec3(clamp(totalColor.x, 0, 1), 
                         clamp(totalColor.y, 0, 1), 
                         clamp(totalColor.z, 0, 1)); 

   output[x + y * resWidth] += totalColor;
}

void allocateGPUScene(const TKSceneData &data, Triangle **dGeomList,
      Triangle ***g_dLightList, int *retGeometryCount, 
      int *retLightCount, Shader **g_dShader) {
   int geometryCount = 0;
   int biggestListSize = 0;
   int lightCount = 0;

   int imgWidth, imgHeight;
   if (data.textureMap.size() > kMaxTextures) {
      printf("Too many textures, max supported is %d\n", kMaxTextures);
      exit(1);
   }

   for (map<string, int>::const_iterator itr = data.textureMap.begin(); 
        itr != data.textureMap.end(); 
        itr++) {

      Tex &curTex = getTexture(itr->second);
      unsigned char *texData = readBMP(itr->first.c_str(), &imgWidth, &imgHeight);

      int imgSize = sizeof(uchar4) * imgWidth * imgHeight;
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);

      hipArray* cu_array;
      hipMallocArray(&cu_array, &channelDesc, imgWidth, imgHeight );

      //copy image to device array cu_array – used as texture mytex on device
      HANDLE_ERROR(hipMemcpyToArray(cu_array, 0, 0, texData, imgSize, hipMemcpyHostToDevice));
      
      curTex.addressMode[0] = hipAddressModeWrap;
      curTex.addressMode[1] = hipAddressModeWrap;
      curTex.filterMode = hipFilterModeLinear;
      curTex.normalized = true; 

      // Bind the array to the texture
      HANDLE_ERROR(hipBindTextureToArray(curTex, cu_array, channelDesc));
   }

   TKTriangle *dTriangleTokens = NULL;
   TKSmoothTriangle *dSmthTriTokens = NULL;

   // Count all the triangles that are also lights
   for (int i = 0; i < data.triangles.size(); i++) { 
      if (data.triangles[i].mod.fin.em) lightCount++;
   }
   for (int i = 0; i < data.smoothTriangles.size(); i++) { 
      if (data.smoothTriangles[i].mod.fin.em) lightCount++;
   }

   int triangleCount = data.triangles.size();
   if (triangleCount > 0) {
      HANDLE_ERROR(hipMalloc(&dTriangleTokens, sizeof(TKTriangle) * triangleCount));
      HANDLE_ERROR(hipMemcpy(dTriangleTokens, &data.triangles[0], 
               sizeof(TKTriangle) * triangleCount, hipMemcpyHostToDevice));
      geometryCount += triangleCount;
      if (triangleCount > biggestListSize) biggestListSize = triangleCount;
   }

   int smoothTriangleCount = data.smoothTriangles.size();
   if (smoothTriangleCount > 0) {
      HANDLE_ERROR(hipMalloc(&dSmthTriTokens, 
               sizeof(TKSmoothTriangle) * smoothTriangleCount));
      HANDLE_ERROR(hipMemcpy(dSmthTriTokens, &data.smoothTriangles[0],
               sizeof(TKSmoothTriangle) * smoothTriangleCount, hipMemcpyHostToDevice));
      geometryCount += smoothTriangleCount;
      if (smoothTriangleCount > biggestListSize) biggestListSize = smoothTriangleCount;
   }

   

   HANDLE_ERROR(hipMalloc(dGeomList, sizeof(Triangle) * geometryCount));
   HANDLE_ERROR(hipMalloc(g_dLightList, sizeof(Triangle *) * lightCount));

   int blockSize = kBlockWidth * kBlockHeight;
   int gridSize = (biggestListSize - 1) / blockSize + 1;
   // Fill up GeomList and LightList with actual objects on the GPU
   initScene<<<gridSize, blockSize>>>(*dGeomList, dTriangleTokens, triangleCount, dSmthTriTokens, smoothTriangleCount);

   hipDeviceSynchronize();
   checkCUDAError("initScene failed");

   if (dTriangleTokens) HANDLE_ERROR(hipFree(dTriangleTokens));
   if (dSmthTriTokens) HANDLE_ERROR(hipFree(dSmthTriTokens));

   *retGeometryCount = geometryCount;
   *retLightCount = lightCount;
}

__global__ void averagePasses(int resWidth, int resHeight, vec3 image[], vec3 outImage[], int passes) {
   int x = blockIdx.x * blockDim.x + threadIdx.x; 
   int y = blockIdx.y * blockDim.y + threadIdx.y; 
   if (x >= resWidth || y >= resHeight) return;

   int idx = y * resWidth + x;

   outImage[idx] = image[idx] * 255.0f / (float)passes;
}

__device__ float getMedian(vec3 list[], int listSize, int axis) {
   int pivot;
   int start = 0;
   int end = listSize;
   int topOfBottom;
   do {
      pivot = (start + end) / 2;
      SWAP(list[pivot], list[end - 1]); 
      topOfBottom = start;
      for (int i = start; i < end - 1; i++) {
         if (list[i][axis] < list[pivot][axis]) {
            SWAP(list[i], list[topOfBottom]);
            topOfBottom++;
         }
      }
      SWAP(list[end - 1], list[topOfBottom]);
      if (topOfBottom < listSize / 2) start = topOfBottom + 1;
      else if (topOfBottom > listSize / 2) end = topOfBottom;
   } while (topOfBottom != listSize / 2);
   return list[pivot][axis];
}

__global__ void medianFilter(int resWidth, int resHeight, vec3 image[]) {
   int x = blockIdx.x * blockDim.x + threadIdx.x; 
   int y = blockIdx.y * blockDim.y + threadIdx.y; 

   if (x >= resWidth || y >= resHeight) return;
   int idx = y * resWidth + x;

   vec3 nearClrs[kMedianPixelAmt * kMedianPixelAmt];
   for (int i = 0; i < kMedianPixelAmt; i++) {
      for (int j = 0; j < kMedianPixelAmt; j++) {
         nearClrs[i * kMedianPixelAmt + j] = image[min(max((y + j - kMedianPixelAmt / 2) * resWidth + x + i - kMedianPixelAmt / 2, 0), resWidth * resHeight - 1)];
      }
   }
   
   image[idx].x = getMedian(nearClrs, kMedianPixelAmt * kMedianPixelAmt, kXAxis);
   image[idx].y = getMedian(nearClrs, kMedianPixelAmt * kMedianPixelAmt, kYAxis);
   image[idx].z = getMedian(nearClrs, kMedianPixelAmt * kMedianPixelAmt, kZAxis);
}

__global__ void blurFilter(int resWidth, int resHeight, vec3 image[]) {
   int x = blockIdx.x * blockDim.x + threadIdx.x; 
   int y = blockIdx.y * blockDim.y + threadIdx.y; 

   if (x >= resWidth || y >= resHeight) return;
   int idx = y * resWidth + x;

   vec3 nearClrs[kMedianPixelAmt * kMedianPixelAmt];
   for (int i = 0; i < kMedianPixelAmt; i++) {
      for (int j = 0; j < kMedianPixelAmt; j++) {
         nearClrs[i * kMedianPixelAmt + j] = image[min(max((y + j - kMedianPixelAmt / 2) * resWidth + x + i - kMedianPixelAmt / 2, 0), resWidth * resHeight - 1)];
      }
   }

   vec3 clr(0.0f);
   for (int i = 0; i < kMedianPixelAmt * kMedianPixelAmt; i++) {
      clr += nearClrs[i];
   }
   image[idx] = clr / (float)(kMedianPixelAmt * kMedianPixelAmt);
}

__global__ void convertToUchar4(int resWidth, int resHeight, vec3 vec3Clrs[], uchar4 output[]) {
   
   int x = blockIdx.x * blockDim.x + threadIdx.x; 
   int y = blockIdx.y * blockDim.y + threadIdx.y; 
   // Flip the image
   int flippedY = resHeight - y - 1; 


   if (x >= resWidth || y >= resHeight) return;

   int idx = y * resWidth + x;
   vec3 clr = vec3Clrs[idx];
   uchar4 convClr;
   convClr.x = clamp(clr.x, 0, 255); convClr.y = clamp(clr.y, 0, 255); 
   convClr.z = clamp(clr.z, 0, 255); convClr.w = 255;
   output[flippedY * resWidth + x] = convClr;
}

RayCache *g_dRayCache;
BVHTree *g_dBvhTree;
hiprandState *g_dRandStates;
vec3 *g_dVec3Out;
vec3 *g_dTempImage;
Shader **g_dShader;
Triangle **g_dLightList;
int g_lightCount;
bool g_isStatic; 
float g_exposureTime;; 
int g_passesPerUpdate; 
float g_timePerUpdate; 

Camera g_camera(vec3(0.0f), vec3(0.0f), vec3(0.0f), vec3(0.0f));

extern "C" void init_kernel(const TKSceneData &data, bool isStatic, int width, int height, 
      float timePerUpdate, int passesPerUpdate, float exposureTime) {

   Triangle *dGeomList; 
   int geometryCount;

   dim3 dimBlock, dimGrid;

   g_isStatic = isStatic;
   g_exposureTime = exposureTime;
   g_passesPerUpdate = passesPerUpdate;
   g_timePerUpdate = timePerUpdate;

   HANDLE_ERROR(hipDeviceSetLimit(hipLimitMallocHeapSize, kGimmeLotsOfMemory));

   TKCamera camTK = *data.camera;
   g_camera = Camera(camTK.pos, camTK.up, camTK.right, 
                 glm::normalize(camTK.lookAt - camTK.pos));

   // Fill the geomList and light list with objects dynamically created on the GPU
   HANDLE_ERROR(hipMalloc(&g_dShader, sizeof(Shader*)));
   HANDLE_ERROR(hipMalloc(&g_dVec3Out , sizeof(vec3) * width * height));
   HANDLE_ERROR(hipMalloc(&g_dTempImage, sizeof(vec3) * width * height));
   HANDLE_ERROR(hipMemset(g_dVec3Out, 0, sizeof(vec3) * width * height));

   allocateGPUScene(data, &dGeomList, &g_dLightList, &geometryCount, &g_lightCount, g_dShader);

   hipDeviceSynchronize();
   checkCUDAError("AllocateGPUScene failed");

   HANDLE_ERROR(hipMalloc(&g_dBvhTree, sizeof(BVHTree)));
   formBVH(dGeomList, geometryCount, g_dBvhTree);

   //formBVH modifies the ordering of geometry so the lights must be gathered afterwards
   setupLights<<<1, 1>>>(dGeomList, geometryCount, g_dLightList);

   HANDLE_ERROR(hipMalloc(&g_dRandStates, sizeof(hiprandState) * kBlockWidth * kBlockHeight));

   dimBlock = dim3(kBlockWidth * kBlockHeight);
   initCurand<<<1, dimBlock>>>(g_dRandStates, kBlockWidth * kBlockHeight);
   HANDLE_ERROR(hipMalloc(&g_dRayCache, sizeof(RayCache) * width* height));

}

extern "C" void launch_kernel(int width, int height, int maxDepth, int pass, uchar4 *dOutput, bool blur, bool median) {

   static float time = 0.0f;
   if (g_exposureTime > 0.0f && time > g_exposureTime) return;

   if (!g_isStatic && pass % g_passesPerUpdate == 0) {
      updateBVH(g_timePerUpdate);
      time += g_timePerUpdate;
   }

   dim3 dimBlock = dim3(kBlockWidth, kBlockHeight);
   dim3 camGrid((width - 1) / kBlockWidth + 1, (height - 1) / kBlockHeight + 1);
   dim3 dimGrid = dim3((kColumnsPerKernel - 1) / kBlockWidth + 1, (kColumnsPerKernel - 1) / kBlockHeight + 1);

   generateCameraRays<<<camGrid, dimBlock>>>(width, height, g_camera, g_dRayCache, g_dRandStates);
   if (pass == 1) hipMemset(g_dVec3Out, 0, sizeof(vec3) * width * height);

   hipDeviceSynchronize();
   for (int depth = 0; depth < maxDepth; depth++) {
      for (int x = 0; x < width ; x += kColumnsPerKernel) {
         for (int y = 0; y < height; y += kRowsPerKernel) {
            rayTrace<<<dimGrid, dimBlock>>>(x, y, width, height,
               g_dBvhTree, g_dLightList, g_lightCount, g_dVec3Out, 
               g_dRandStates, g_dRayCache, depth);
         }
      }
   }
   hipDeviceSynchronize();
   checkCUDAError("rayTrace kernel failed");

   dimBlock = dim3(kBlockWidth, kBlockHeight);
   dimGrid = dim3((width - 1) / kBlockWidth + 1, (height - 1) / kBlockHeight + 1);

   // If in interactive mode
   if (g_exposureTime < 0.0f) {
      averagePasses<<<dimGrid, dimBlock>>>(width, height, g_dVec3Out, g_dTempImage, pass); 

   // If in photo mode
   } else {
      averagePasses<<<dimGrid, dimBlock>>>(width, height, g_dVec3Out, g_dTempImage, g_exposureTime * g_passesPerUpdate / g_timePerUpdate); 
   }
   hipDeviceSynchronize();
   checkCUDAError("averagePasses kernel failed");

   if (median) {
      medianFilter<<<dimGrid, dimBlock>>>(width, height, g_dTempImage); 
      hipDeviceSynchronize();
      checkCUDAError("medianFilter kernel failed");
   }

   if (blur) {
      blurFilter<<<dimGrid, dimBlock>>>(width, height, g_dTempImage); 
      hipDeviceSynchronize();
      checkCUDAError("blurFilter kernel failed");
   }

   convertToUchar4<<<dimGrid, dimBlock>>>(width, height, g_dTempImage, dOutput); 
   hipDeviceSynchronize();
   checkCUDAError("convertToUchar4 kernel failed");
}

void translateCamera(glm::vec3 dir) {
   g_camera.pos += dir.x * glm::normalize(g_camera.right);
   g_camera.pos += dir.z * g_camera.lookAtDir;
   g_camera.pos += dir.y * g_camera.up;
}

void rotateCameraSideways(float angle) {
   mat4 rot = glm::rotate(mat4(1.0f), angle, g_camera.up);
   g_camera.right = vec3(rot * vec4(g_camera.right.x, 
                                    g_camera.right.y, 
                                    g_camera.right.z, 0.0f));
   g_camera.lookAtDir = vec3(rot * vec4(g_camera.lookAtDir.x, 
                                    g_camera.lookAtDir.y, 
                                    g_camera.lookAtDir.z, 0.0f));
}
void rotateCameraVertically(float angle) {
   glm::mat4 rot = glm::rotate(glm::mat4(1.0f), angle, g_camera.right);
   g_camera.up = vec3(rot * vec4(g_camera.up.x, 
                                    g_camera.up.y, 
                                    g_camera.up.z, 0.0f));
   g_camera.lookAtDir = vec3(rot * vec4(g_camera.lookAtDir.x, 
                                    g_camera.lookAtDir.y, 
                                    g_camera.lookAtDir.z, 0.0f));
}

void kernelGetImage(uchar4 *dImage, uchar4 *hostImage, int width, int height) {
   HANDLE_ERROR(hipMemcpy(hostImage, dImage, 
            sizeof(uchar4) * width * height, hipMemcpyDeviceToHost));
   hipDeviceSynchronize();
}
